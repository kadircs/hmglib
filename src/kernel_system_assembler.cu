#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>

#include "system_assembler.h"
//#include "generic_system_adapter.h"
#include "kernel_system_assembler.h"

__global__ void create_gaussian_kernel_system_assembler_object_kernel(struct gaussian_kernel_system_assembler** assem, double regularization)
{
	(*assem) = new gaussian_kernel_system_assembler();
	(*assem)->regularization = regularization;
}

void create_gaussian_kernel_system_assembler_object(struct gaussian_kernel_system_assembler** assem, double regularization)
{
	create_gaussian_kernel_system_assembler_object_kernel<<<1,1>>>(assem, regularization);
	hipDeviceSynchronize();
	checkCUDAError("create_gaussian_kernel_system_assembler_object");
}


__global__ void destroy_gaussian_kernel_system_assembler_object_kernel(struct gaussian_kernel_system_assembler** assem)
{
	delete *assem;
}

void destroy_gaussian_kernel_system_assembler_object(struct gaussian_kernel_system_assembler** assem)
{
	destroy_gaussian_kernel_system_assembler_object_kernel<<<1,1>>>(assem);
	hipDeviceSynchronize();
 	checkCUDAError("destroy_gaussian_kernel_system_assembler_object");
}



__global__ void create_matern_kernel_system_assembler_object_kernel(struct matern_kernel_system_assembler** assem, double regularization)
{
	(*assem) = new matern_kernel_system_assembler();
	(*assem)->regularization = regularization;
}

void create_matern_kernel_system_assembler_object(struct matern_kernel_system_assembler** assem, double regularization)
{
	create_matern_kernel_system_assembler_object_kernel<<<1,1>>>(assem, regularization);
	hipDeviceSynchronize();
	checkCUDAError("create_matern_kernel_system_assembler_object");
}


__global__ void destroy_matern_kernel_system_assembler_object_kernel(struct matern_kernel_system_assembler** assem)
{
	delete *assem;
}

void destroy_matern_kernel_system_assembler_object(struct matern_kernel_system_assembler** assem)
{
	destroy_matern_kernel_system_assembler_object_kernel<<<1,1>>>(assem);
	hipDeviceSynchronize();
 	checkCUDAError("destroy_matern_kernel_system_assembler_object");
}



