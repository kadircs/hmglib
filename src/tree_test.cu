#include "hip/hip_runtime.h"
// Copyright (C) 2016 Peter Zaspel
//
// This file is part of hmglib.
//
// hmglib is free software: you can redistribute it and/or modify it under the
// terms of the GNU Lesser General Public License as published by the Free
// Software Foundation, either version 3 of the License, or (at your option) any
// later version.
//
// hmglib is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
// details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with hmglib.  If not, see <http://www.gnu.org/licenses/>.

#include <stdio.h>
#include "morton.h"
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <hiprand.h>
#include <thrust/sort.h>
#include "tree.h"
#include "linear_algebra.h"
#include <thrust/inner_product.h>
#include "helper.h"


int main( int argc, char* argv[])
{

//	int dim = 3;
//	int bits = 20;
	int dim = 2;
	int bits = 32;

//	int point_count=16;
//	int point_count=20;
	int point_count = atoi(argv[1]);
//	int point_count=4000000;
//	int point_count=29000000;

	if (argc!=6)
	{
		printf("./tree_test <N> <k> <c_leaf> <exponent of epsilon> <eta>\n");
		return 0;
	}

	// allocating memory for point_count coordinates in dim dimensions
	double** coords_d;
	coords_d = new double*[dim];
	for (int d = 0; d < dim; d++)
	{
		hipMalloc((void**)&(coords_d[d]), point_count*sizeof(double));
		checkCUDAError("hipMalloc");
	}

	// allocating memory for extremal values per dimension
	double* max_per_dim_d;
	hipMalloc((void**)&max_per_dim_d, dim*sizeof(double));
	double* min_per_dim_d;
	hipMalloc((void**)&min_per_dim_d, dim*sizeof(double));

	// generating device pointer that holds the dimension-wise access
	double** coords_device;
	hipMalloc((void**)&(coords_device), dim*sizeof(double*));
	hipMemcpy(coords_device, coords_d, dim*sizeof(double*), hipMemcpyHostToDevice);
	
	// allocationg memory for morton codes
	uint64_t* code_d;
	hipMalloc((void**)&code_d, point_count*sizeof(uint64_t));
	checkCUDAError("hipMalloc");

	// setting up data strcture for point set
	struct point_set* points_d;
	hipMalloc((void**)&points_d, sizeof(struct point_set));
	init_point_set<<<1,1>>>(points_d, coords_device, dim, max_per_dim_d, min_per_dim_d, point_count);

	// setting up data structure for morton code
	struct morton_code* morton_d;
	hipMalloc((void**)&morton_d, sizeof(struct morton_code));
	init_morton_code<<<1,1>>>(morton_d, code_d, dim, bits, point_count);


	// generate random points
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	for (int d = 0; d < dim; d++ )
	{
		hiprandGenerateUniformDouble(gen, coords_d[d], point_count);
	}
	hiprandDestroyGenerator(gen);



//	set_2d_test_set<<<1,16>>>(points_d);

//	set_3d_test_set<<<1,64>>>(points_d);



	// compute extremal values for the point set
	compute_minmax(points_d);
	
	// calculate GPU thread configuration	
	int block_size = 512;
	int grid_size = (point_count + (block_size - 1)) / block_size;



	// generate morton codes
	TIME_start;
	get_morton_code<<<grid_size, block_size>>>(points_d, morton_d);
	TIME_stop("get_morton_3d");
	checkCUDAError("get_morton_code");

//	print_points_with_morton_codes(points_d, morton_d);


	// find ordering of points following Z curve
	uint64_t* order;
	hipMalloc((void**)&order, point_count*sizeof(uint64_t));
	get_morton_ordering(points_d, morton_d, order);

//	print_points_with_morton_codes(points_d, morton_d);

	// reorder points following the morton code order
//	TIME_start;
	reorder_point_set(points_d, order);
//	TIME_stop("reorder_point_set");

//	double eta=10.0;
//	double eta=0.5;
	double eta=atof(argv[5]);
	int max_level=50; // DEBUG
//	int c_leaf=1024;
	int c_leaf=atoi(argv[3]);
//	int c_leaf=4;

	struct work_item root_h;
	root_h.set1_l = 0;
	root_h.set1_u = point_count - 1;
	root_h.set2_l = 0;
	root_h.set2_u = point_count - 1;

	int mat_vec_data_count = 0;  // will be filled with the size number of mat_vec_data entries

/*
	int max_elements_in_array = -1; // TODO
	int max_elements_in_mat_vec_data_array = point_count*7; // TODO

	struct work_item* mat_vec_data;
	hipMalloc((void**)&mat_vec_data, max_elements_in_mat_vec_data_array*sizeof(struct work_item));

	TIME_start;
	traverse_with_dynamic_arrays(root_h, mat_vec_data, &mat_vec_data_count, morton_d, morton_d, points_d, points_d, eta, max_level, c_leaf, max_elements_in_array);
	TIME_stop("traverse_with_arrays");

	printf("mat_vec_data_count: %d\n", mat_vec_data_count);
	print_work_items(mat_vec_data, mat_vec_data_count);

	hipFree(mat_vec_data);
*/


	int max_elements_in_array = -1; // TODO
	int max_elements_in_mat_vec_data_array = -1; // TODO

	struct work_item** mat_vec_data = new struct work_item*[1];
	int mat_vec_data_array_size = 1048576;
	hipMalloc((void**)mat_vec_data, mat_vec_data_array_size*sizeof(struct work_item));

	TIME_start;
	traverse_with_dynamic_arrays_dynamic_output(root_h, mat_vec_data, &mat_vec_data_count, &mat_vec_data_array_size, morton_d, morton_d, points_d, points_d, eta, max_level, c_leaf, max_elements_in_array);
	TIME_stop("traverse_with_arrays");

//	printf("mat_vec_data_count: %d\n", mat_vec_data_count);
//	print_work_items(*mat_vec_data, mat_vec_data_count);
//	char file_name[1000];
//	sprintf(file_name, "mat_vec_data.txt");
//	write_work_items(file_name, *mat_vec_data, mat_vec_data_count);

	double* x;
	double* y;
	double* y_test;
	hipMalloc((void**)&x, point_count*sizeof(double));
	hipMalloc((void**)&y, point_count*sizeof(double));
	hipMalloc((void**)&y_test, point_count*sizeof(double));

	// generate random vector x
	hiprandGenerator_t vec_gen;
	hiprandCreateGenerator(&vec_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandGenerateUniformDouble(vec_gen, x, point_count);
	hiprandDestroyGenerator(vec_gen);

//	TIME_start;
//	struct work_item test_mat_vec_data;
//	test_mat_vec_data.set1_l = 0;
//	test_mat_vec_data.set1_u = point_count - 1;
//	test_mat_vec_data.set2_l = 0;
//	test_mat_vec_data.set2_u = point_count - 1;
//	double* test_matrix;
//	hipMalloc((void**)&test_matrix, point_count*point_count*sizeof(double));
//	checkCUDAError("hipMalloc");
//	int block_size1 = 512;
//	fill_matrix<<<(point_count*point_count + (block_size1 - 1)) / block_size1, block_size1>>>(test_matrix, test_mat_vec_data, points_d, points_d, point_count, point_count);
//	hipDeviceSynchronize();
//	checkCUDAError("fill_matrix");
//    hipblasStatus_t stat;
//    hipblasHandle_t handle;
//    stat = hipblasCreate(&handle);
//	// matrix-vector-product
//	double one;
//	double zero;
//	one = 1.0;
//	zero = 0.0;
//	stat = hipblasDgemv(handle, HIPBLAS_OP_N, point_count, point_count, &one, test_matrix, point_count, x, 1, &zero, y_test, 1);
//	if (stat!=HIPBLAS_STATUS_SUCCESS)
//	{
//		printf("dgemv did not succeed...\n");
//		exit(1);
//	}
//    hipblasDestroy(handle);
//    TIME_stop("dense_mvp");
//
//    hipFree(test_matrix);

	int k = atoi(argv[2]);

	double epsilon;
	epsilon = pow(10.0, atoi(argv[4]));

	mat_vec_data_info mat_vec_info;
	organize_mat_vec_data(*mat_vec_data, mat_vec_data_count, &mat_vec_info);

	printf("dort\n");
	TIME_start;
	sequential_h_matrix_mvp(x, y, *mat_vec_data, &mat_vec_info, points_d, points_d, point_count, eta, epsilon, k);
	TIME_stop("sequential_h_matrix");

	thrust::device_ptr<double> y_ptr(y);
	thrust::device_ptr<double> y_test_ptr(y_test);

//	printf("x,y, y_test\n");
//	print_double(x, point_count);
//	print_double(y, point_count);
//	print_double(y_test, point_count);

	thrust::transform(y_test_ptr, y_test_ptr+point_count, y_ptr, y_test_ptr, thrust::minus<double>());
	double error = sqrt(thrust::inner_product(y_test_ptr, y_test_ptr+point_count, y_test_ptr, 0.0));

	printf("Error: %le\n", error);
//
	hipFree(y_test);

	hipFree(x);
	hipFree(y);

	hipFree(*mat_vec_data);




/*	int tree_work_queue_size = 100*point_count; // DEBUG
	int mat_vec_work_queue_size = 100*point_count ; // DEBUG

	struct work_queue **tree_work_queue_h = new struct work_queue*[max_level];
	struct work_queue **tree_work_queue = new struct work_queue*[max_level];

	for (int l=0; l<max_level; l++)
	{
		allocate_work_queue(&(tree_work_queue_h[l]), &(tree_work_queue[l]), tree_work_queue_size);
		// calculate GPU thread configuration
		int invalidate_block_size = 512;
		int invalidate_grid_size = (tree_work_queue_size + (invalidate_block_size - 1)) / invalidate_block_size;
		invalidate_work_items<<<invalidate_grid_size,invalidate_block_size>>>(tree_work_queue_h[l]->data, tree_work_queue_size);
	}
	
	struct work_queue tmp;
	hipMemcpy(&tmp, tree_work_queue[0], sizeof(struct work_queue), hipMemcpyDeviceToHost);
	printf("work size: %d\n", tmp.queue_size);


	struct work_queue **tree_work_queue_dev;
	hipMalloc((void**)&tree_work_queue_dev, max_level*sizeof(struct work_queue*));
	
	hipMemcpy(tree_work_queue_dev, tree_work_queue, max_level*sizeof(struct work_queue*), hipMemcpyHostToDevice);
	
	struct work_queue *mat_vec_work_queue_h;
	struct work_queue *mat_vec_work_queue;

	allocate_work_queue(&mat_vec_work_queue_h, &mat_vec_work_queue, mat_vec_work_queue_size);
		
	init_tree_work_queue_root<<<1,1>>>(tree_work_queue_dev, 0, point_count-1, 0, point_count-1);
	hipDeviceSynchronize();
	checkCUDAError("init_tree_work_queue_root");

	traverse_single_single_queue<<<1, 4>>>(mat_vec_work_queue, tree_work_queue_dev, morton_d, morton_d, points_d, points_d, eta, max_level, c_leaf);
//	traverse<<<grid_size, block_size>>>(mat_vec_work_queue, tree_work_queue_dev, morton_d, morton_d, points_d, points_d, eta, max_level, c_leaf);
	hipDeviceSynchronize();
	checkCUDAError("traverse");

	hipMemcpy(mat_vec_work_queue_h, mat_vec_work_queue, sizeof(struct work_queue), hipMemcpyDeviceToHost);
	checkCUDAError("memcpy");

	struct work_item* items = new struct work_item[mat_vec_work_queue_size];

	printf("memcpy: %p %p %lu\n",items, mat_vec_work_queue_h->data, mat_vec_work_queue_size*sizeof(struct work_queue));

	int real_mat_vec_work_queue_size = mat_vec_work_queue_h->tail - mat_vec_work_queue_h->head;

	printf("data %p\n",mat_vec_work_queue_h->data);
	printf("head %p\n",mat_vec_work_queue_h->head);
	printf("tail %p\n",mat_vec_work_queue_h->tail);
	printf("size %ld\n",mat_vec_work_queue_h->tail-mat_vec_work_queue_h->head);

	hipMemcpy(items, mat_vec_work_queue_h->data, real_mat_vec_work_queue_size*sizeof(struct work_item), hipMemcpyDeviceToHost);
	checkCUDAError("memcpy");

	for (int it=0; it<real_mat_vec_work_queue_size; it++)
	{
		printf("item %d: %d %d %d %d, %d\n",it, items[it].set1_l, items[it].set1_u, items[it].set2_l, items[it].set2_u, items[it].work_type);
	}
	delete [] items;

	delete_work_queue(mat_vec_work_queue_h, mat_vec_work_queue);

	hipFree(tree_work_queue_dev);

	for (int l=0; l<max_level; l++)
		delete_work_queue(tree_work_queue_h[l], tree_work_queue[l]);
	
*/

	
	
	
//	// print ordered points
//	print_points(points_d);

//	// write points to file
//	char file_name[2000];
//	sprintf(file_name,"points.dat");
//	write_points(points_d,file_name);
	

	hipFree(order);

	// freeing memory for morton codes
	hipFree(code_d);

	// freeing coordinates memory
	for (int d = 0; d < dim; d++)
	{
		hipFree(coords_d[d]);
	}
	delete [] coords_d;

}
